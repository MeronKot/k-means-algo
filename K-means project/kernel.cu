#include "hip/hip_runtime.h"
#include "k-means_header.h"

void error(Point *dev_points, Point *dev_clusters, float *dev_dis)
{
	hipFree(dev_points);
	hipFree(dev_clusters);
	hipFree(dev_dis);
}

__global__ void distanceKernel(Point *points,Point *clusterPoints,float *dis, int n,int procId,int k)
{
    float x = 0,y = 0;
	int p = threadIdx.x;
	
	if( (p >= (procId * k)) && (p < (procId * k) + k) )
	{
		p = p % k;
		for(int i = 0 ; i < n ; i++)
		{
			x = clusterPoints[p].x - points[i].x;
			y = clusterPoints[p].y - points[i].y;
			dis[p*n + i] = sqrt(x*x + y*y);
		}
	}
}

hipError_t distanceWithCuda(Point *points, Point *clusterPoints,  float **dis , int n, int k,int procId,int numprocs)
{
    Point *dev_points = 0;
    Point *dev_clusters = 0;
	float *dev_dis = 0;
	hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        error(dev_points,dev_clusters,dev_dis);
    }
    // Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_points, n * sizeof(Point));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        error(dev_points,dev_clusters,dev_dis);
    }

	cudaStatus = hipMalloc((void**)&dev_clusters, k * sizeof(Point));
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        error(dev_points,dev_clusters,dev_dis);
    }
	
	cudaStatus = hipMalloc((void**)&dev_dis, k * n * sizeof(float));

	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        error(dev_points,dev_clusters,dev_dis);
    }

    // Copy input vectors from host memory to GPU buffers.

    cudaStatus = hipMemcpy(dev_points, points, n * sizeof(Point), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        error(dev_points,dev_clusters,dev_dis);
    }

	cudaStatus = hipMemcpy(dev_clusters, clusterPoints, k * sizeof(Point), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		error(dev_points,dev_clusters,dev_dis);
    }

	//cudaStatus = hipMemcpy2D(dev_dis,pitch,dis,pitch,k,n,hipMemcpyHostToDevice);
	for(int i = 0 ; i < k ; i++)
		cudaStatus = hipMemcpy(&dev_dis[i * n], dis[i],n * sizeof(float), hipMemcpyHostToDevice);
	
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        error(dev_points,dev_clusters,dev_dis);
	}

    // Launch a kernel on the GPU with one thread for each element.
	distanceKernel<<<1, k*numprocs>>>(dev_points,dev_clusters,dev_dis,n,procId - 1,k);
	  
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "distanceKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        error(dev_points,dev_clusters,dev_dis);
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching distanceKernel!\n", cudaStatus);
         error(dev_points,dev_clusters,dev_dis);
    }

    // Copy output vector from GPU buffer to host memory.
    for (int i = 0 ; i < k ; i++)
		cudaStatus = hipMemcpy(dis[i], &dev_dis[i * n], n * sizeof(float), hipMemcpyDeviceToHost);
	
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        error(dev_points,dev_clusters,dev_dis);
    }
   
	hipFree(dev_points);
	hipFree(dev_clusters);
	hipFree(dev_dis);
	return cudaStatus;
}


